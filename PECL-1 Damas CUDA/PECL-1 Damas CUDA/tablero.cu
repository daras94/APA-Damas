#include "hip/hip_runtime.h"
﻿#include "tablero.cuh"

/*
	Recupera las carrasteristicas nesesarias para realizar la configuracion del tablero.

	- devian	 = recibe un entero con la el id de la GPU que se va a usar para realizar la configuracion.
	- deviceProp = recibe un struct para almacenar las carrasteristicas de la GPU.
	- myConfGpu  = truckt pasado por referencia para almacenar las informacion de la gpu ge nos interesa.
*/
void getCofigPlay(int devian, hipDeviceProp_t *deviceProp, info_gpu *myConfGpu) {
	hipSetDevice(devian);														// Establecemos con que GPU queremos realizar la configuracion en funcion de disponer.
	hipGetDeviceProperties(deviceProp, devian);
	myConfGpu->numThreadMaxPerSM = deviceProp->maxThreadsPerMultiProcessor;
	myConfGpu->numThreadMasPerBlock = deviceProp->maxThreadsPerBlock;
	myConfGpu->numRegPerBlock = deviceProp->regsPerBlock;
	myConfGpu->sharedMemPerBlock = deviceProp->sharedMemPerBlock;
	for (int i = 0; i < NUM_DIMENSION; i++) {
		myConfGpu->maxDimGridSize[i] = deviceProp->maxGridSize[i];
		myConfGpu->maxDimThreadBlock[i] = deviceProp->maxThreadsDim[i];
	}
}

/*
	Medodo que genera 3 posbles configuraciones de la dimeciones de la trageta grafica
	en funcion de la carateristicas de la GPU que posea el usuario.
		
		- devProp       = puntero a struc el cual contiene la informacion de la GPU es 
					      de la arquitectura de CUDA.
		- myConfGpu     = struck declarado en la cabecera de esta clase el cul usamos 
		                  para almacenar informacion concreta de la GPU.
		- deviceCurrent = Id de la GPU que actualmente tenemos selecionada como principal.
*/
double setGpuForPlayAuto(hipDeviceProp_t *devProp,  info_gpu *myConfGpu, int deviceCurrent) {
	double *dimTamblero, numThread;
	int  gpuOpc;
	do {
		system("cls");
		cout << "/***************************************************************************************/" << endl;
		cout << "/*  +--> " << ANSI_COLOR_CYAN "Menu de configuracion de partida:" ANSI_COLOR_RESET << setw(47) << "*/" << endl;
		cout << "/*  ---------------------------------------------------------------------------------  */" << endl;
		cout << "/*" << setw(87) << "*/" << endl;
			string modelGPU = devProp->name;
			cout << "/*  " ANSI_COLOR_MAGENTA "GPU " << deviceCurrent << ANSI_COLOR_RESET ") - " << modelGPU << setw(76 - modelGPU.length()) << "*/" << endl;
			cout << "/*  ---------------------------------------------------------------------------------  */" << endl;
			numThread = myConfGpu->maxDimThreadBlock[1];
			dimTamblero = new double[NUM_DIMENSION] { 2, 4, 8 };
			for (int i = 0; i < 3; i++) {
				dimTamblero[i] = numThread / dimTamblero[i];
				cout << "/*\t" << right << ANSI_COLOR_MAGENTA << (i + 1) << ANSI_COLOR_RESET ") - Disponible tablero de juego de " << ANSI_COLOR_GREEN << (dimTamblero[i] / TAM_TESELA) << "x" << (dimTamblero[i] / TAM_TESELA) << ANSI_COLOR_RESET " " << dimTamblero[i] << " Threads" << setw(40) << "*/" << endl;
			}
			cout << "/*  ---------------------------------------------------------------------------------  */" << endl;
			cout << "/*  - " << ANSI_COLOR_RED "AVISO: " ANSI_COLOR_RESET "Selecione un tablero para la fiesta de CUDA." << setw(35) << "*/" << endl;
		cout << "/***************************************************************************************/" << endl;
		cout << " - Selecione una opcion para juegar (" ANSI_COLOR_GREEN "0 para salir de la configuracion" ANSI_COLOR_RESET "): ";
		cin >> gpuOpc;		// Entrada de texto por teclado.
		if (gpuOpc != 0 && (gpuOpc < 0 || gpuOpc > 3)) {
			ERROR_MSS("Error opcion de juego introducida no es valida.");
		}
	} while (gpuOpc != 0 && (gpuOpc < 0 || gpuOpc > 3));
	return dimTamblero[gpuOpc - 1];
}

/*
	Medodo que genera 3 posbles configuraciones de la dimeciones de la trageta grafica
	en funcion de la carateristicas de la GPU que posea el usuario.

		- devProp       = puntero a struc el cual contiene la informacion de la GPU es
						  de la arquitectura de CUDA.
		- myConfGpu     = struck declarado en la cabecera de esta clase el cul usamos
						  para almacenar informacion concreta de la GPU.
		- deviceCurrent = Id de la GPU que actualmente tenemos selecionada como principal.
*/
double setGpuForPlayManual(hipDeviceProp_t *devProp, info_gpu *myConfGpu, int deviceCurrent) {
	int imput, cont = 0; double dim = 1;
	system("cls");
	cout << "/***************************************************************************************/" << endl;
	cout << "/*  +--> " << ANSI_COLOR_CYAN "Menu de configuracion de partida:" ANSI_COLOR_RESET << setw(47) << "*/" << endl;
	cout << "/*  ---------------------------------------------------------------------------------  */" << endl;
	cout << "/*" << setw(87) << "*/" << endl;
	string modelGPU = devProp->name;
	cout << "/*  " ANSI_COLOR_MAGENTA "GPU " << deviceCurrent << ANSI_COLOR_RESET ") - " << modelGPU << setw(76 - modelGPU.length()) << "*/" << endl;
	cout << "/*  ---------------------------------------------------------------------------------  */" << endl;
	cout << "/*  - " << ANSI_COLOR_RED "AVISO: " ANSI_COLOR_RESET "Cuidado con la configuracion quien rompe CUDA lo sufre." << setw(35) << "*/" << endl;
	cout << "/***************************************************************************************/" << endl;
	teclado:
	cout << " - Introduca el numero de " << ((cont == 0)? "filas" : "Columnas") << " (" ANSI_COLOR_GREEN "0 para salir" ANSI_COLOR_RESET "): ";
	cin >> imput;		// Entrada de texto por teclado.
	if (cont == 0 && imput != 0) {
		dim = dim * imput;
		cont++;
		goto teclado;
	}
	return dim;
}

/*
	Establece la dificultade la partida.
*/
int setDificultad() {
	int  dificultad;
	do {
		system("cls");
		cout << "/***************************************************************************************/" << endl;
		cout << "/*  +--> " << ANSI_COLOR_CYAN "Nivel de dificulta de partida:" ANSI_COLOR_RESET << setw(52) << "*/" << endl;
		cout << "/*  ---------------------------------------------------------------------------------  */" << endl;
		cout << "/*" << setw(87) << "*/" << endl;
		string niveles[NIVEL_DIFICULTAD] = {"Muy Facil", "Facil", "Normal", "Avanzado", "Experto"};
		for (int  i = 0; i < NIVEL_DIFICULTAD; i++) {
			cout << "/*\t" ANSI_COLOR_MAGENTA << (i + 1) << ANSI_COLOR_RESET ") - "  << ANSI_COLOR_RESET << niveles[i] << setw(60 - niveles[i].length()) << "*/" << endl;
		}
		cout << "/*" << setw(87) << "*/" << endl;
		cout << "/***************************************************************************************/" << endl;
		cout << " - Selecione dificultad del juego (" ANSI_COLOR_GREEN "0 para salir de la partida" ANSI_COLOR_RESET "): ";
		cin >> dificultad;		// Entrada de texto por teclado.
		if (dificultad != 0 && (dificultad < 0 && dificultad > NIVEL_DIFICULTAD)) {
			ERROR_MSS("Error opcion de juego introducida no es valida.");
		}
	} while (dificultad != 0 && (dificultad < 0 && dificultad > NIVEL_DIFICULTAD));
	return dificultad;
}

/* 
	Generamos el tablero con un números de bonbas aleatorios en función de la dificultad.
*/
long *generarTablero(double numThread, int dificultad) {
	long row = 0, col = 0, *tablero = new long[(int)numThread];
	int numRowFicha = log2(numThread / TAM_TESELA);			// El numero de fichas para cada jugador en funcion de las dimensiones del tablero.
	srand(time(NULL));
	for (int i = 0; i < numThread; i++) { 
		row = i / ((int)numThread / TAM_TESELA);			// Calculamos la columna 
		col = ((row % 2) == 0)? 1 : 0;						// Calculamos el desplazamiento de la fichas en la colocacion.
		int bonba = rand() % dificultad;				    // Gennera Bombas en funcion de las dificultad selecionada.
		tablero[i] = (((col + i) % 2) == 0)? (row < numRowFicha)? 11 + bonba : POS_TAB_JUEGO_EMPTY : (row >= (numRowFicha * (numRowFicha - 1)))? 22 + bonba : POS_TAB_JUEGO_EMPTY;
	}
	return tablero;
}

/* 
	Función que imprime el número de columnas que va a tener el tablero
	para que sea más facil elegir piezas.
*/
void imprimirColumnas(double numThread) {
	for (int i = 0; i < (numThread / TAM_TESELA); i++) {
		cout << ((i == 0) ? setw(12) : (i < 9) ? setw(3) : setw(3.5)) << i + 1;
	}
	cout << "" << endl;
	for (int i = 0; i < (numThread / TAM_TESELA); i++) {
		cout << ((i == 0)? setw(12) : setw(3)) << "|";
	}
	cout << "" << endl;
}

//Imprimimos el tablero
void imprimirTablero(long *tablero, double numThread) {
	imprimirColumnas(numThread);
	for (int i = 0; i < numThread / TAM_TESELA; i++) {
		cout << setw(4) << i+1 << setw(3) << "-" << setw(3) << "";
		for (int k = 0; k < numThread/TAM_TESELA; k++) {								// Damos color en función del número imprimir
			int background = ((i + k) % 2 == 0) ? COLOR_GRIS : COLOR_NEGRO;			// Color que contrulle el tablero.
			long bloque = tablero[i * ((int)numThread / TAM_TESELA) + k];
			//if (bloque < NUM_FICHAS) {												// Calculamos el color de la casilla.
				int color = COLOR_TABLERO(background, (new int[NUM_FICHAS] {background, COLOR_ROJO, COLOR_AZUL, COLOR_VERDE, COLOR_PURPURA, COLOR_AMARILLO, COLOR_AGUAMARINA, COLOR_PURPURA_LIGHT})[bloque % 10]); 
				SetConsoleTextAttribute(GetStdHandle(STD_OUTPUT_HANDLE), color);
			//} 
				cout << " " << (((bloque - (bloque % 10)) > POS_TAB_JUEGO_EMPTY)? "#" : "O") << " ";
		}
		cout << ANSI_COLOR_RESET "" << endl;
	}
}

/*
	Medodo que se encarga de gestionar la partida, salvar la partida con persistencia
	y lanzar el kernel adecuado que el jugador aya selecionado.
*/ 
void playDamas(int typeKernel, double numThread, info_gpu *myConfGpu, int dificultad) {
	long *tablero = generarTablero(numThread, dificultad);
	long cont = 0;
	string input = { NULL };
	do {
		system("cls");
		cout << "/***************************************************************************************/" << endl;
		cout << "/*  +--> " << ANSI_COLOR_CYAN "Tablero de juego, Turno de juego de ficha: " ANSI_COLOR_RESET << (new string[2]{"#","O"})[(cont % 2 == 0)? 0 : 1] << setw(36) << "*/" << endl;
		cout << "/*  ---------------------------------------------------------------------------------  */" << endl;
		cout << "  " << setw(87) << "  " << endl;
		imprimirTablero(tablero, numThread);
		cout << "  " << setw(87) << "  " << endl;
		cout << "/*  -----------------------------------------------------------------------------------  */" << endl;
		cout << "/*  - " << ANSI_COLOR_RED "AVISO: " ANSI_COLOR_RESET "Jugada con el formato X:Y:D (X = column, Y = row, " << setw(28) << " */" << endl;
		cout << "/*    " << setw(72) << " D = (10 = sup-izq, 20 = inf-izq, 11 = sup-dech, 21 = inf-dech)). "			   << setw(13) << " */" << endl;
		cout << "/*****************************************************************************************/" << endl;
		teclado:
		cout << " - Realice su jugada (" ANSI_COLOR_GREEN "0 para salir de la partida s para guardar la partida." ANSI_COLOR_RESET "): ";
		cin >> input;															// Entrada de texto por teclado.
		smatch match;
		regex  reg_expre{R"(\d{1,2}:\d{1,2}:(1|2){1}(0|1){1})"};				// Epresion regular para las filas y columnas.
		bool found = regex_match(input, match, reg_expre);						// Coparacion que busca un expresion de tipo fila:columna:direccion
		if (found) {
			int *jugada = getRowAndColumn(input, numThread);
			if (sizeof(jugada) < NUM_DIMENSION_TAB) {
				ERROR_MSS("Error en la columna o fila introducida.");
				goto teclado;
			} else {															// Inbocamos al metodo de lanzamiento de los kernels
				bool error_play = launchKernel(typeKernel, numThread, tablero, jugada);
				if (error_play) {
					ERROR_MSS("El movimento realizado no es valido.");
					goto teclado;
				}
				cont++;
				system("pause");
			}
		} else {
			switch ((char)&input) {
				case 's':									// Para la persistencia desde la partida.
					
					break;
				default:									// Carraterees no validos
					if (input != "0") {
						ERROR_MSS("Error carrater o movimiento introducido no valido no valida.");
						goto teclado;
					}
					break;
			}
		}
	} while (input != "0");
}

/*
	Metodo Que inboca el kernel segun el tipo de kernel que se quiera ejecutar.

		- typeKernel = Entero que indica tipo de kernel a lanzar x Block, Mem_Shared or Bloques y Mem_Global.
		- mumThread  = recibe el numeroi de thread para realizar la configuracion de juego.
		- tablero	 = Recibe el tablero de juego generado por el host
		- jugada	 = Recibe la jugada realizada por algun jugador.
*/
bool launchKernel(int typeKernel, double numThread, long* tablero, int* jugada) {
	bool isErrorJugada = false;
	switch (typeKernel) {
		case 1:		// Memoria Compartida Con Colesencia y Teselada.
			launchKernelMemShared(numThread, tablero, jugada, isErrorJugada);
			break;
		case 2:		// Por Bloques.

			break;
		case 3:		// Por Bloques Con Memoria Compartida.

			break;
	}
	return isErrorJugada;
}

/*
	Realiza el Separado de los valores de la jugada pasada en el formato C:F:D 
	(C = Columna, F = Fila, D = Direcion) y debuelve la jugada como un array 
	de enteros. 
*/
int *getRowAndColumn(string jug, double numThread) {
	string delimiter = ":", aux = jug + ":";
	int pos = 0, cont = 0, *rowCol = new int[NUM_DIMENSION_TAB];
	bool isNotErrorColRow = true;
	while ((pos = aux.find(delimiter)) != string::npos && isNotErrorColRow) {
		int token = stoi(aux.substr(0, pos));
		if (isNotErrorColRow = (token > 0 && token <= (numThread / TAM_TESELA))) {
			rowCol[cont] = token;
		}
		aux.erase(0, pos + delimiter.length());
		cont++;
	}

	return rowCol;
}

