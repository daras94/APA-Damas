#include "UtilGPU.cuh"

/*
	Transfiere el contenoido de la mmemoria de la GPU a la memoria, Los Argumentos 
	son los siguientes y retorna el codigo de error en caso de error
 		- c    =  puntero a entero en el que devolvemos.
		- dev  =  puntero a enteros usado para tranferir a la GPU.
		- size =  devuelve el tama�o del valor a transferir.
*/
void setCudaMemcpyToHost(long*& c, long*& dev, int size) {
	// Copy output vector from GPU buffer to host memory.
	hipError_t cudaStatus = hipMemcpy(c, dev, size * sizeof(long), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		ERROR_MSS("Fallo el la operacion hipMemcpy Device to Host!!");
		fprintf(stderr, hipGetErrorString(cudaStatus));
		goto Error;
	}
	// Saltamos a erroe y liberamos la memoria.
	return;
	Error:
		hipFree(dev);
		cout << endl;
		system("pause");
}

void setCudaMemcpyToDevice(long*& c, long*& dev, int size) {
	// Copy output vector from GPU buffer to host memory.
	hipError_t cudaStatus = hipMemcpy(c, dev, size * sizeof(long), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		ERROR_MSS("Fallo el la operacion hipMemcpy Host to Device!!");
		fprintf(stderr, hipGetErrorString(cudaStatus));
		goto Error;
	}
	// Saltamos a erroe y liberamos la memoria.
	return;
Error:
	hipFree(dev);
	cout << endl;
	system("pause");
}

/*
	Asignamos memoria a la variable pasada por parametro para pasarlas a la GPU, , Los Argumentos 
	son los siguientes y retorna el codigo de error en caso de error
		- c    =  puntero a entero en el que devolvemos.
		- dev  =  puntero a enteros usado para tranferir a la GPU.
		- size =  devuelve el tama�o del valor a transferir.
*/
void setCudaMalloc(long*& dev, int size) {
	// Allocate GPU buffers for three vectors (two input, one output)    .
	hipError_t cudaStatus = hipMalloc((void**)&dev, size * sizeof(long));
	if (cudaStatus != hipSuccess) {
		ERROR_MSS("Fallo el la operacion hipMalloc !!");
		fprintf(stderr, hipGetErrorString(cudaStatus));
		goto Error;
	}
	// Saltamos a error y liberamos la memoria.
	return;
Error:
	hipFree(&dev);
	cout << endl;
	system("pause");
}