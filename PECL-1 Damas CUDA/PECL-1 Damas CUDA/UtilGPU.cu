#include "UtilGPU.cuh"

/*
	Transfiere el contenoido de la mmemoria de la GPU a la memoria, Los Argumentos 
	son los siguientes y retorna el codigo de error en caso de error
 		- c    =  puntero a entero en el que devolvemos.
		- dev  =  puntero a enteros usado para tranferir a la GPU.
		- size =  devuelve el tama�o del valor a transferir.
*/
hipError_t setCudaMemcpy(int *c,  int *dev, unsigned int size) {
	// Copy output vector from GPU buffer to host memory.
	hipError_t cudaStatus = hipMemcpy(c, dev, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		ERROR_MSS("Fallo el la operacion hipMemcpy !!");
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	// Saltamos a erro y liberamos la memoria.
	Error:
		hipFree(dev);
		return cudaStatus;
}

/*
	Asignamos memoria a la variable pasada por parametro para pasarlas a la GPU, , Los Argumentos 
	son los siguientes y retorna el codigo de error en caso de error
		- c    =  puntero a entero en el que devolvemos.
		- dev  =  puntero a enteros usado para tranferir a la GPU.
		- size =  devuelve el tama�o del valor a transferir.
*/
hipError_t setCudaMalloc(int *dev, unsigned int size) {
	// Allocate GPU buffers for three vectors (two input, one output)    .
	hipError_t cudaStatus = hipMalloc((void**)&dev, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		ERROR_MSS("Fallo el la operacion hipMalloc !!");
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	// Saltamos a erro y liberamos la memoria.
Error:
	hipFree(dev);
	return cudaStatus;
}